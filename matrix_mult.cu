/*****************************************************************************
 * File:        matrix_mult.cu
 *
 * Run:         ./matrix_mult
 *****************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


// __global__
void basic_mult(float* matrix_A, float* matrix_B, float* out, int size)
{
  for (int output_col = 0; output_col < size; output_col++){
    for (int row = 0; row < size; row++){
      float val = 0;
      for (int col = 0; col <= row; col++){
        val += matrix_A[row*size + col]*matrix_B[col*size + output_col];
      }
      out[row*size + output_col] = val;
    }
  }
}

void printMatrix(float* matrix, int matrix_size){
  for(int i = 0; i < matrix_size; i++){
    for(int j = 0; j < matrix_size; j++){
      printf(" %f", matrix[matrix_size*i + j]);
    }
    printf("\n");
  }
}


bool multiply(int matrix_size, float sparsity)
{
    unsigned int bytes = matrix_size * matrix_size * sizeof(float);
    float* h_A, *h_B, *h_out;
    float* d_A, *d_B, *d_out;
    double throughput;

    // allocate host memory
    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);
    h_out = (float*)malloc(bytes);

    // init inputs
    for (int i = 0; i < matrix_size; i++){
        for (int j = 0; j <= i; j++){
            h_A[i*matrix_size + j] = rand()/(float)RAND_MAX < sparsity ? rand() / (float)RAND_MAX : 0;
            h_B[i*matrix_size + j] = rand()/(float)RAND_MAX < sparsity ? rand() / (float)RAND_MAX : 0;
        }
    }

    basic_mult(h_A, h_B, h_out, matrix_size);

    printf("Matrix A:\n ");
    printMatrix(h_A, matrix_size);
    printf("Matrix B:\n ");
    printMatrix(h_B, matrix_size);
    printf("Result:\n ");
    printMatrix(h_out, matrix_size);


    // allocate device memory
    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_out, bytes);
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // launch Kernel
    // printf("\n %s\n", "Launch Kernel....");
    // Timer timer;
    // if (kernel_type == "vanilla") {
    //     // basic 2d conv
    //     //todo: define block and grid size
    //     dim3 dimBlock(16, 16, 1);
	  //     dim3 dimGrid(1, 1, 1);
    //     printf("block dim: %d x %d x %d\n", dimBlock.x, dimBlock.y, dimBlock.z);
    //     printf("grid dim: %d x %d x %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
    //
    //     //todo: warmup
    //
    //     startTime(&timer);
    //     //todo: launch kernel on device and test performance, get results
    //     convolution2D<<<dimBlock, dimGrid>>>(d_in, d_out, width, height, channels, kernel_width);
	  //     cudaDeviceSynchronize();
    //     stopTime(&timer);
    //     double flopsPerMatrixMul = 2.0 * static_cast<double>(kernel_width) * \
    //                                 static_cast<double>(kernel_width) * static_cast<double>(kernel_width); \
    //     double numMatrixMul = width*height*channels;
    //     throughput = (numMatrixMul*flopsPerMatrixMul * 1.0e-9f) / (elapsedTime(timer) / 1000.0f);
    // }
    // else if (kernel_type == "shared_mem") {
    //     //  2d conv on shared mem
    //     //todo: define block and grid size
    //
    //     dim3 dimBlock(16, 16, 1);
    //     dim3 dimGrid(16, 16, 1);
    //
    //     printf("block dim: %d x %d x %d\n", dimBlock.x, dimBlock.y, dimBlock.z);
    //     printf("grid dim: %d x %d x %d\n", dimGrid.x, dimGrid.y, dimGrid.z);
    //
    //     //todo: warmup
    //     convolution2D_sharedmem<<<dimBlock, dimGrid>>>(d_in, d_out, width, height, channels, kernel_width);
    //
    //     startTime(&timer);
    //     convolution2D_sharedmem<<<dimBlock, dimGrid>>>(d_in, d_out, width, height, channels, kernel_width);
    //     cudaDeviceSynchronize();
    //     stopTime(&timer);
    //
    //     double flopsPerMatrixMul = 2.0 * static_cast<double>(kernel_width) * \
    //                                 static_cast<double>(kernel_width) * static_cast<double>(kernel_width); \
    //     double numMatrixMul = width*height*channels;
    //     throughput = (numMatrixMul*flopsPerMatrixMul * 1.0e-9f) / (elapsedTime(timer) / 1000.0f);
    // }
    // todo: bonus optimization
    /*
    else if (kernel_type == "optimized") {
    }
    */

    // result in CPU
    // float* cpu_out = (float*)malloc(bytes);
    // printf("\nCalculating in CPU...\n");
    // verify(h_in, cpu_out, h_kernel, width, height, channels, kernel_width);
    //
    // int precision = 8;
    // double threshold = 1e-8 * channels*width*height;
    // double diff = 0.0;
    //
    // //todo: compare kernel result with CPU result
    // float* gpu_out = (float*)malloc(bytes);
    // CUDA_CHECK(cudaMemcpy(gpu_out, d_out, bytes, cudaMemcpyDeviceToHost));
    // printf("%s\n", cudaGetErrorString(cudaGetLastError()));
    //
    // for(int i = 0; i < bytes/sizeof(float); i++){
    //   diff += abs(cpu_out[i] - gpu_out[i]);
    // }
    //
    // printf("Kernel width is %d \n", kernel_width);
    //
    // //todo: getting result
    // printf("[Kernel %s] Throughput = %.4f GB/s, Time = %.5f ms\n",
    //     kernel_type, throughput, elapsedTime(timer)*1000);
    // printf("Error : %.*f (threshold: %f)\n", precision, (double)diff, threshold);

    // todo:
    // free memory (both device and host mem)

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);
    free(h_A);
    free(h_B);
    free(h_out);
    // free(cpu_out);
    // free(gpu_out);

    return true;
}

/*
 * Argument:
 *      "--width=<N>"       : Specify the number of width of input image (default: 1024)
 *      "--height=<N>"      : Specify the number of height of input image (default: 2048)
 *      "--channel=<N>"     : Specify the number of channels of input image (default: 1, <= 3)
 *      "--filter=<N>"      : Specify the number of filter width for convolution (default: 5)
*/

int main(int argc, char** argv)
{
    printf("[Multiplying matrices...]\n\n");

    int matrix_size = 10;
    float sparsity = 0.7;

    // if (checkCmdLineFlag(argc, (const char **)argv, "width")) {
    //     width = getCmdLineArgumentInt(argc, (const char **)argv, "width");
    // }
    // if (checkCmdLineFlag(argc, (const char **)argv, "height")) {
    //     height = getCmdLineArgumentInt(argc, (const char **)argv, "height");
    // }
    // if (checkCmdLineFlag(argc, (const char **)argv, "channels")) {
    //     channels = getCmdLineArgumentInt(argc, (const char **)argv, "channels");
    // }
    // if (checkCmdLineFlag(argc, (const char **)argv, "filter")) {
    //     kernel_width = getCmdLineArgumentInt(argc, (const char **)argv, "filter");
    // }

    int dev = 0;
    hipSetDevice(dev);

    multiply(matrix_size, sparsity);

    // bool result;
    // result = compute(width, height, channels, kernel_width, "vanilla");
    // printf(result ? "Test PASSED\n" : "Test FAILED!\n");
    //
    //
    // result = compute(width, height, channels, kernel_width, "shared_mem");
    // printf(result ? "Test PASSED\n" : "Test FAILED!\n");


    //todo: bonus
    //result = compute(width, height, channels, kernel_width, "optimized");
    //printf(result ? "Test PASSED\n" : "Test FAILED!\n");
    hipDeviceReset();

    return 0;
}
